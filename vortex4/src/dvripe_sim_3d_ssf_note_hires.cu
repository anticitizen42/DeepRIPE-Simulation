#include "hip/hip_runtime.h"
/******************************************************************************
 * dvripe_sim_3d_ssf_note_hires.cu
 *
 * A single-file CUDA + cuFFT + OpenGL + GLFW program that:
 *   1) Runs a 3D Split-Step Fourier focusing Schrödinger PDE with a "Note"
 *      potential (trap/pattern).
 *   2) Uses a higher resolution domain (128x128x64) for more detail.
 *   3) Displays the amplitude slice (z=NZ/2) in a larger window with
 *      pixel zoom for a bigger, more defined output.
 *
 * Build on Windows (example):
 *   nvcc dvripe_sim_3d_ssf_note_hires.cu -o dvripe_sim_3d_ssf_note_hires.exe ^
        -I"C:\vcpkg\installed\x64-windows\include" ^
        -L"C:\vcpkg\installed\x64-windows\lib" ^
        -lcufft -lglew32 -lglfw3dll -lopengl32
 *
 * Build on Linux (example):
 *   nvcc dvripe_sim_3d_ssf_note_hires.cu -o dvripe_sim_3d_ssf_note_hires \
 *       -lcufft -lGLEW -lglfw -lGL
 *
 * Notes:
 *   - NX=128, NY=128, NZ=64 => 1,048,576 cells, heavier on GPU.
 *   - The window is set to (NX*UPSCALE) x (NY*UPSCALE), plus glPixelZoom.
 *   - If performance is slow or GPU memory is insufficient, reduce domain size.
 ******************************************************************************/

#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define _USE_MATH_DEFINES
#include <math.h>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// Prevent GLFW from including gl.h
#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

//---------------------------------------------------------------------------
// Domain & PDE parameters
//---------------------------------------------------------------------------
static const int   NX    = 128;     // bigger resolution in x
static const int   NY    = 128;     // bigger resolution in y
static const int   NZ    = 64;      // somewhat large in z

static const float DX    = 0.1f;    // spatial step
static const float DT    = 0.0001f; // time step
static const float DVAL  = 0.5f;    // dispersion
static const float GVAL  = 2.0f;    // focusing strength
static const float NOTE  = 1.0f;    // "Note" potential parameter
static const int   STEPS_PER_FRAME = 1;

// We’ll upscale the slice display by 6x
static const int   UPSCALE = 6;

//---------------------------------------------------------------------------
// GPU float2 helpers
//---------------------------------------------------------------------------
__device__ inline float2 cMul(const float2& a, const float2& b)
{
    // (a.x + i a.y)*(b.x + i b.y)
    return make_float2(a.x*b.x - a.y*b.y,
                       a.x*b.y + a.y*b.x);
}

__device__ inline float cAbs2(const float2& a)
{
    return a.x*a.x + a.y*a.y;
}

__device__ inline float2 cExp(float phase)
{
    return make_float2(cosf(phase), sinf(phase));
}

//---------------------------------------------------------------------------
// "Note" Potential: harmonic-like for demonstration
//---------------------------------------------------------------------------
__device__ float computeNotePotential(int i, int j, int k,
                                      int nx, int ny, int nz,
                                      float noteParam)
{
    float cx = nx/2.0f;
    float cy = ny/2.0f;
    float cz = nz/2.0f;

    float x = i - cx;
    float y = j - cy;
    float z = k - cz;
    float r2 = x*x + y*y + z*z;

    // V = 0.5 * noteParam * r^2
    float V = 0.5f * noteParam * r2;
    return V;
}

//---------------------------------------------------------------------------
// We'll apply the Note potential in real space as an extra half-step
//    ψ <- ψ * exp(-i V_note dt/2)
//---------------------------------------------------------------------------
__global__ void applyNotePotentialHalfStep(float2* psi,
                                           int nx, int ny, int nz,
                                           float dt, float noteParam)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    if(i<nx && j<ny && k<nz)
    {
        int idx = (k*ny + j)*nx + i;
        float2 val = psi[idx];

        float V = computeNotePotential(i,j,k, nx,ny,nz, noteParam);
        float phase = - V * dt * 0.5f;

        float2 e = make_float2(cosf(phase), sinf(phase));
        float2 out;
        out.x = val.x*e.x - val.y*e.y;
        out.y = val.x*e.y + val.y*e.x;
        psi[idx] = out;
    }
}

//---------------------------------------------------------------------------
// Nonlinear half-step: focusing
//    ψ <- ψ * exp(i * [-G|ψ|^2] dt/2)
//---------------------------------------------------------------------------
__global__ void nonlinearHalfStep(float2* psi,
                                  int nx, int ny, int nz,
                                  float dt, float G)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    if(i<nx && j<ny && k<nz)
    {
        int idx = (k*ny + j)*nx + i;
        float2 val = psi[idx];
        float amp2 = cAbs2(val);

        // focusing => phase = - G amp2 dt/2
        float phase = - G*amp2 * (dt*0.5f);

        float2 e = make_float2(cosf(phase), sinf(phase));
        float2 out;
        out.x = val.x*e.x - val.y*e.y;
        out.y = val.x*e.y + val.y*e.x;
        psi[idx] = out;
    }
}

//---------------------------------------------------------------------------
// Linear step in Fourier space
//---------------------------------------------------------------------------
__global__ void linearStepKernel(float2* psi_hat,
                                 int nx, int ny, int nz,
                                 float dt, float D,
                                 float dkx, float dky, float dkz)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < nx*ny*nz)
    {
        int z = idx / (nx*ny);
        int r = idx % (nx*ny);
        int y = r / nx;
        int x = r % nx;

        int sx = (x < nx/2) ? x : (x - nx);
        int sy = (y < ny/2) ? y : (y - ny);
        int sz = (z < nz/2) ? z : (z - nz);

        float kx = sx*dkx;
        float ky = sy*dky;
        float kz = sz*dkz;
        float k2 = kx*kx + ky*ky + kz*kz;

        float phase = -D*k2*dt;
        float2 e = make_float2(cosf(phase), sinf(phase));

        float2 val = psi_hat[idx];
        float2 out;
        out.x = val.x*e.x - val.y*e.y;
        out.y = val.x*e.y + val.y*e.x;
        psi_hat[idx] = out;
    }
}

// scale after inverse FFT
__global__ void scaleKernel(float2* data, float s, int n)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < n)
    {
        data[idx].x *= s;
        data[idx].y *= s;
    }
}

//---------------------------------------------------------------------------
// We'll fill a PBO with the amplitude slice z=NZ/2
// Each pixel is RGBA (uchar4). We'll do a grayscale mapping of amplitude.
//---------------------------------------------------------------------------
__global__ void fillSliceKernel(uchar4* pbo,
                                const float2* psi,
                                int nx, int ny, int nz, int sliceZ)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if(i<nx && j<ny)
    {
        int idx3D = (sliceZ*ny + j)*nx + i;
        float2 val = psi[idx3D];
        float amp = sqrtf(val.x*val.x + val.y*val.y);

        // arbitrary scaling
        float scale = amp*10.0f;
        if(scale>1.0f) scale=1.0f;
        unsigned char c = (unsigned char)(scale*255.0f);

        int idx2D = j*nx + i;
        pbo[idx2D] = make_uchar4(c, c, c, 255);
    }
}

//---------------------------------------------------------------------------
// We'll store the slice in a Pixel Buffer (PBO)
//---------------------------------------------------------------------------
static GLuint pboID=0;
static struct hipGraphicsResource* pboResource=nullptr;

bool createSlicePBO(int width, int height)
{
    glGenBuffers(1, &pboID);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboID);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, width*height*4, nullptr, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    hipError_t err = hipGraphicsGLRegisterBuffer(&pboResource,
                                                   pboID,
                                                   hipGraphicsRegisterFlagsWriteDiscard);
    if(err != hipSuccess)
    {
        std::cerr << "hipGraphicsGLRegisterBuffer failed: "
                  << hipGetErrorString(err) << std::endl;
        return false;
    }
    return true;
}

void renderSlicePBO(int width, int height)
{
    // We'll do a glDrawPixels with pixel zoom
    glPixelZoom((float)UPSCALE, (float)UPSCALE);

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboID);
    glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // reset pixel zoom
    glPixelZoom(1.0f, 1.0f);
}

//---------------------------------------------------------------------------
// Main
//---------------------------------------------------------------------------
int main()
{
    // 1) Create a bigger window
    if(!glfwInit())
    {
        std::cerr << "Failed to init GLFW\n";
        return -1;
    }
    int winW = NX*UPSCALE;
    int winH = NY*UPSCALE;
    GLFWwindow* window = glfwCreateWindow(winW, winH,
                                          "DVRIPE 3D SSF + Note Potential - HiRes",
                                          nullptr, nullptr);
    if(!window)
    {
        std::cerr << "Failed to create GLFW window\n";
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    // 2) Init GLEW
    GLenum glewErr = glewInit();
    if(glewErr != GLEW_OK)
    {
        std::cerr << "GLEW error: " << glewGetErrorString(glewErr) << std::endl;
        glfwDestroyWindow(window);
        glfwTerminate();
        return -1;
    }

    // 3) Create PBO for slice
    if(!createSlicePBO(NX, NY))
    {
        std::cerr << "Failed to create slice PBO\n";
        return -1;
    }

    // 4) Orthographic 2D for drawing the slice
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    // from (0..NX, 0..NY)
    glOrtho(0, NX, 0, NY, -1,1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    // 5) Allocate field on host
    int n = NX*NY*NZ;
    std::vector<float2> psiHost(n);

    // swirl initial
    float cx = NX/2.0f;
    float cy = NY/2.0f;
    float cz = NZ/2.0f;
    for(int z=0; z<NZ; z++)
    {
        for(int y=0; y<NY; y++)
        {
            for(int x=0; x<NX; x++)
            {
                int idx=(z*NY + y)*NX + x;
                float dx_ = x - cx;
                float dy_ = y - cy;
                float dz_ = z - cz;
                float r2  = dx_*dx_ + dy_*dy_ + dz_*dz_;
                float amp = expf(-r2/(NX*DX*0.3f));
                float th  = atan2f(dy_, dx_);
                psiHost[idx] = make_float2(amp*cosf(th), amp*sinf(th));
            }
        }
    }

    // 6) Allocate on device
    float2* psiDev=nullptr;
    hipMalloc(&psiDev, n*sizeof(float2));
    hipMemcpy(psiDev, psiHost.data(), n*sizeof(float2), hipMemcpyHostToDevice);

    // PDE config
    hipfftHandle plan;
    if(hipfftPlan3d(&plan, NZ, NY, NX, HIPFFT_C2C) != HIPFFT_SUCCESS)
    {
        std::cerr << "hipfftPlan3d failed!\n";
        return -1;
    }

    float dkx = 2.0f*M_PI/(NX*DX);
    float dky = 2.0f*M_PI/(NY*DX);
    float dkz = 2.0f*M_PI/(NZ*DX);

    dim3 threads(8,8,8);
    dim3 blocks((NX+threads.x-1)/threads.x,
                (NY+threads.y-1)/threads.y,
                (NZ+threads.z-1)/threads.z);

    int copyThreads=256;
    int copyBlocks=(n+copyThreads-1)/copyThreads;

    dim3 sliceThreads(16,16);
    dim3 sliceBlocks((NX+sliceThreads.x-1)/sliceThreads.x,
                     (NY+sliceThreads.y-1)/sliceThreads.y);

    // Device function to combine "nonlinear" + "note potential" in real space
    // We'll do them sequentially for clarity
    auto doNonlinearNoteHalfStep = [&](float2* psiDev)
    {
        // 1) Nonlinear half-step
        nonlinearHalfStep<<<blocks, threads>>>(psiDev, NX, NY, NZ, DT, GVAL);
        hipDeviceSynchronize();

        // 2) Note potential half-step
        applyNotePotentialHalfStep<<<blocks, threads>>>(psiDev,
                                                        NX, NY, NZ,
                                                        DT, NOTE);
        hipDeviceSynchronize();
    };

    while(!glfwWindowShouldClose(window))
    {
        // PDE steps
        for(int step=0; step<STEPS_PER_FRAME; step++)
        {
            // Nonlinear + note half-step
            doNonlinearNoteHalfStep(psiDev);

            // Linear step in Fourier space
            hipfftExecC2C(plan, (hipfftComplex*)psiDev, (hipfftComplex*)psiDev, HIPFFT_FORWARD);

            linearStepKernel<<<copyBlocks, copyThreads>>>(psiDev,
                                                          NX, NY, NZ,
                                                          DT, DVAL,
                                                          dkx, dky, dkz);
            hipDeviceSynchronize();

            hipfftExecC2C(plan, (hipfftComplex*)psiDev, (hipfftComplex*)psiDev, HIPFFT_BACKWARD);

            // scale
            float scaleVal = 1.0f/(float)(NX*NY*NZ);
            scaleKernel<<<copyBlocks, copyThreads>>>(psiDev, scaleVal, n);
            hipDeviceSynchronize();

            // Nonlinear + note half-step again
            doNonlinearNoteHalfStep(psiDev);
        }

        // fill slice z=NZ/2
        hipGraphicsMapResources(1, &pboResource, 0);
        size_t numBytes=0;
        uchar4* d_pbo=nullptr;
        hipGraphicsResourceGetMappedPointer((void**)&d_pbo, &numBytes, pboResource);

        fillSliceKernel<<<sliceBlocks, sliceThreads>>>(d_pbo,
                                                       psiDev,
                                                       NX, NY, NZ,
                                                       NZ/2);
        hipDeviceSynchronize();
        hipGraphicsUnmapResources(1, &pboResource, 0);

        // Render
        glClear(GL_COLOR_BUFFER_BIT);
        glRasterPos2f(0,0);
        renderSlicePBO(NX, NY);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // Cleanup
    hipfftDestroy(plan);
    hipFree(psiDev);

    hipGraphicsUnregisterResource(pboResource);
    glDeleteBuffers(1, &pboID);

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
