#include "hip/hip_runtime.h"
/******************************************************************************
 * dvripe_sim_3d_polarwavelet_realtime.cu
 *
 * Demonstrates a 3D focusing PDE that collapses into a "dot," plus a
 * dual-radius polar discretization + wavelet transform in angle.
 *
 * We show three panels in one window:
 *   1) Left : The 2D slice amplitude (z = NZ/2).
 *   2) Center: Wavelet scalogram (scale vs. angle) for radius R1.
 *   3) Right : Wavelet scalogram (scale vs. angle) for radius R2.
 *
 * Build on Windows (example):
 *   nvcc dvripe_sim_3d_polarwavelet_realtime.cu -o dvripe_sim_3d_polarwavelet_realtime.exe ^
 *       -I"C:\vcpkg\installed\x64-windows\include" ^
 *       -L"C:\vcpkg\installed\x64-windows\lib" ^
 *       -lglew32 -lglfw3dll -lopengl32
 *
 *   (Adjust library names as needed, e.g. -lglfw3 for static link.)
 *
 * Build on Linux (example):
 *   nvcc dvripe_sim_3d_polarwavelet_realtime.cu -o dvripe_sim_3d_polarwavelet_realtime \
 *       -lGLEW -lglfw -lGL
 ******************************************************************************/

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// Ensure M_PI is available
#define _USE_MATH_DEFINES
#include <math.h>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// Prevent GLFW from including OpenGL headers
#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>

// Include GLEW after GLFW
#include <GL/glew.h>

#include <cuda_gl_interop.h>

// -----------------------------------------------------------------------------
// 3D PDE Parameters
// -----------------------------------------------------------------------------
static const int NX = 64;  // grid size in x
static const int NY = 64;  // grid size in y
static const int NZ = 64;  // grid size in z

static const float DX = 0.1f;    // spatial step
static const float DT = 0.0001f; // time step
static const float D  = 1.0f;    // diffusion/dispersion coefficient
static const float G  = 1.0f;    // focusing strength

static const int STEPS_PER_FRAME = 1; // PDE steps per rendered frame

// We'll show the slice z=NZ/2 in the left panel
static const int SLICE_Z = NZ/2;

// -----------------------------------------------------------------------------
// "Dual" radii for polar wavelet analysis
// -----------------------------------------------------------------------------
static const float R1 = 3.0f;   // inner radius
static const float R2 = 6.0f;   // outer radius
static const int   NUM_ANGLES = 512; // fine angular resolution
static const int   NUM_SCALES = 64;  // wavelet scales
static const float W0 = 6.0f;       // Morlet wavelet carrier freq

// -----------------------------------------------------------------------------
// 3D complex field
// -----------------------------------------------------------------------------
struct Complex3D {
    float x; // real
    float y; // imag
};

__device__ inline float cAbs2(const Complex3D& c)
{
    return c.x*c.x + c.y*c.y;
}

// -----------------------------------------------------------------------------
// PDE: ∂ψ/∂t = i [ D ∇²ψ - G |ψ|^2 ψ ]
// Focusing because of the -G term
// -----------------------------------------------------------------------------
__global__ void pdeUpdateKernel3D(Complex3D* psi, Complex3D* psiNew,
                                  int nx, int ny, int nz,
                                  float dx, float dt,
                                  float Dval, float Gval)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // x
    int j = blockIdx.y * blockDim.y + threadIdx.y; // y
    int k = blockIdx.z * blockDim.z + threadIdx.z; // z

    if(i < nx && j < ny && k < nz)
    {
        // Flatten 3D -> 1D
        int idx = (k * ny + j)*nx + i;

        auto wrap = [&](int c, int maxVal){
            if(c < 0) c += maxVal;
            if(c >= maxVal) c -= maxVal;
            return c;
        };
        int ip = wrap(i+1, nx);
        int im = wrap(i-1, nx);
        int jp = wrap(j+1, ny);
        int jm = wrap(j-1, ny);
        int kp = wrap(k+1, nz);
        int km = wrap(k-1, nz);

        int idx_ip = (k*ny + j)*nx + ip;
        int idx_im = (k*ny + j)*nx + im;
        int idx_jp = (k*ny + jp)*nx + i;
        int idx_jm = (k*ny + jm)*nx + i;
        int idx_kp = (kp*ny + j)*nx + i;
        int idx_km = (km*ny + j)*nx + i;

        // Laplacian in 3D
        Complex3D lap;
        lap.x = ( psi[idx_ip].x + psi[idx_im].x
                + psi[idx_jp].x + psi[idx_jm].x
                + psi[idx_kp].x + psi[idx_km].x
                - 6.0f*psi[idx].x ) / (dx*dx);
        lap.y = ( psi[idx_ip].y + psi[idx_im].y
                + psi[idx_jp].y + psi[idx_jm].y
                + psi[idx_kp].y + psi[idx_km].y
                - 6.0f*psi[idx].y ) / (dx*dx);

        // i * D * lap
        Complex3D linear;
        linear.x = -lap.y * Dval; // multiply by i => -y
        linear.y =  lap.x * Dval; //                 x

        // i * [-G|ψ|^2] * ψ
        float amp2 = cAbs2(psi[idx]);
        Complex3D nonlin;
        nonlin.x = -psi[idx].y; // multiply by i
        nonlin.y =  psi[idx].x;
        float factor = -Gval * amp2;  // minus sign for focusing
        nonlin.x *= factor;
        nonlin.y *= factor;

        // dpsi/dt
        Complex3D dpsi_dt;
        dpsi_dt.x = linear.x + nonlin.x;
        dpsi_dt.y = linear.y + nonlin.y;

        // Euler step
        psiNew[idx].x = psi[idx].x + dt*dpsi_dt.x;
        psiNew[idx].y = psi[idx].y + dt*dpsi_dt.y;
    }
}

// Copy kernel
__global__ void copyKernel3D(Complex3D* psi, Complex3D* psiNew, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n) {
        psi[idx] = psiNew[idx];
    }
}

// -----------------------------------------------------------------------------
// Initialize field: wide Gaussian
// -----------------------------------------------------------------------------
void initializeField3D(std::vector<Complex3D>& psiHost, int nx, int ny, int nz, float dx)
{
    float cx = nx/2.0f;
    float cy = ny/2.0f;
    float cz = nz/2.0f;

    for(int k=0; k<nz; k++){
        for(int j=0; j<ny; j++){
            for(int i=0; i<nx; i++){
                int idx = (k*ny + j)*nx + i;
                float x = i - cx;
                float y = j - cy;
                float z = k - cz;
                float r2 = x*x + y*y + z*z;

                // wide Gaussian
                float amplitude = expf(-r2 / (nx*dx*0.3f));
                psiHost[idx].x = amplitude; // purely real
                psiHost[idx].y = 0.0f;
            }
        }
    }
}

// -----------------------------------------------------------------------------
// 2D slice kernel: fill a PBO with amplitude from z=SLICE_Z
// -----------------------------------------------------------------------------
__global__ void fillSliceKernel(uchar4* pbo,
                                const Complex3D* psi,
                                int nx, int ny, int nz, int sliceZ)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // x
    int j = blockIdx.y * blockDim.y + threadIdx.y; // y

    if(i < nx && j < ny)
    {
        int idx3D = (sliceZ*ny + j)*nx + i;
        int idx2D = j*nx + i;

        float amp = sqrtf(psi[idx3D].x*psi[idx3D].x + psi[idx3D].y*psi[idx3D].y);

        // grayscale
        float val = amp * 10.0f;
        if(val>1.0f) val=1.0f;
        unsigned char c = (unsigned char)(val*255.0f);

        pbo[idx2D].x = c;
        pbo[idx2D].y = c;
        pbo[idx2D].z = c;
        pbo[idx2D].w = 255;
    }
}

// -----------------------------------------------------------------------------
// Morlet wavelet transform in angle
// -----------------------------------------------------------------------------
static inline float morletReal(float t, float s, float w0)
{
    float x = t/s;
    float gauss = expf(-0.5f*x*x);
    return gauss * cosf(w0*x);
}
static inline float morletImag(float t, float s, float w0)
{
    float x = t/s;
    float gauss = expf(-0.5f*x*x);
    return gauss * sinf(w0*x);
}

// We do a naive O(N^2 * numScales) transform
void computeMorletWavelet1D(const std::vector<float>& angleSignal,
                            int N, int numScales,
                            float scaleMin, float scaleMax, float w0,
                            std::vector<float>& waveletScalogram)
{
    for(int sIdx=0; sIdx<numScales; sIdx++)
    {
        float fraction = (float)sIdx/(float)(numScales-1);
        float scale = scaleMin * powf(scaleMax/scaleMin, fraction);

        for(int tau=0; tau<N; tau++)
        {
            float realPart=0.0f, imagPart=0.0f;
            for(int k=0; k<N; k++)
            {
                int idx = (tau + k) % N;
                float x = (float)k;

                float wR = morletReal(x, scale, w0);
                float wI = morletImag(x, scale, w0);

                realPart += angleSignal[idx]*wR;
                imagPart += angleSignal[idx]*wI;
            }
            float mag = sqrtf(realPart*realPart + imagPart*imagPart);
            waveletScalogram[sIdx*N + tau] = mag;
        }
    }
}

// -----------------------------------------------------------------------------
// Sample amplitude around a circle in the slice z=SLICE_Z
// (cx, cy) is the center in that slice
// -----------------------------------------------------------------------------
void sampleAmplitudeCircle(const std::vector<Complex3D>& psiHost,
                           int nx, int ny, int nz, int sliceZ,
                           float cx, float cy, float radius,
                           int numAngles,
                           std::vector<float>& angleSignal)
{
    for(int i=0; i<numAngles; i++)
    {
        float theta = 2.0f*(float)M_PI*(float)i/(float)numAngles;
        float xCoord = cx + radius*cosf(theta);
        float yCoord = cy + radius*sinf(theta);

        int xi = (int)roundf(xCoord);
        int yi = (int)roundf(yCoord);

        // wrap or clamp
        if(xi<0) xi += nx;
        if(yi<0) yi += ny;
        xi = xi % nx;
        yi = yi % ny;

        int idx3D = (sliceZ*ny + yi)*nx + xi;
        float amp = sqrtf(psiHost[idx3D].x*psiHost[idx3D].x + psiHost[idx3D].y*psiHost[idx3D].y);
        angleSignal[i] = amp;
    }
}

// -----------------------------------------------------------------------------
// CUDA-OpenGL interop
// -----------------------------------------------------------------------------
static GLuint pboID;
static struct hipGraphicsResource* pboResource;

// We'll have TWO wavelet textures: waveletTex1 (for R1) and waveletTex2 (for R2)
static GLuint waveletTex1;
static GLuint waveletTex2;

bool createPBO(int width, int height)
{
    glGenBuffers(1, &pboID);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboID);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, width*height*4, nullptr, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    hipError_t err = hipGraphicsGLRegisterBuffer(&pboResource, pboID, hipGraphicsRegisterFlagsWriteDiscard);
    if(err != hipSuccess) {
        std::cerr << "hipGraphicsGLRegisterBuffer failed: " << hipGetErrorString(err) << std::endl;
        return false;
    }
    return true;
}

bool createWaveletTexture(GLuint& texID, int width, int height)
{
    glGenTextures(1, &texID);
    glBindTexture(GL_TEXTURE_2D, texID);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA,
                 width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    glBindTexture(GL_TEXTURE_2D, 0);
    return true;
}

void updateWaveletTexture(GLuint texID,
                          const std::vector<float>& waveletData,
                          int numScales, int numAngles)
{
    std::vector<unsigned char> texCPU(numScales*numAngles*4);

    float maxVal = 0.0f;
    for(auto v : waveletData){
        if(v>maxVal) maxVal=v;
    }
    if(maxVal < 1e-9f) maxVal=1e-9f;

    for(int s=0; s<numScales; s++){
        for(int a=0; a<numAngles; a++){
            float val = waveletData[s*numAngles + a]/maxVal;
            if(val>1.0f) val=1.0f;
            unsigned char c = (unsigned char)(val*255.0f);

            int idx = (s*numAngles + a)*4;
            texCPU[idx+0] = c;
            texCPU[idx+1] = c;
            texCPU[idx+2] = c;
            texCPU[idx+3] = 255;
        }
    }

    glBindTexture(GL_TEXTURE_2D, texID);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0,
                    numAngles, numScales,
                    GL_RGBA, GL_UNSIGNED_BYTE, texCPU.data());
    glBindTexture(GL_TEXTURE_2D, 0);
}

void drawTexturedQuad(GLuint texID, int x, int y, int w, int h,
                      int windowW, int windowH)
{
    glViewport(x, y, w, h);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0,1,0,1,-1,1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, texID);

    glBegin(GL_QUADS);
    glTexCoord2f(0,0); glVertex2f(0,0);
    glTexCoord2f(1,0); glVertex2f(1,0);
    glTexCoord2f(1,1); glVertex2f(1,1);
    glTexCoord2f(0,1); glVertex2f(0,1);
    glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);
    glDisable(GL_TEXTURE_2D);
}

// -----------------------------------------------------------------------------
// Main
// -----------------------------------------------------------------------------
int main()
{
    if(!glfwInit()){
        std::cerr << "Failed to init GLFW\n";
        return -1;
    }

    // We want 3 panels: left PDE slice, center wavelet R1, right wavelet R2
    // We'll make the window 3*NX wide, NY high
    int winW = NX*3;
    int winH = NY;

    GLFWwindow* window = glfwCreateWindow(winW, winH, "3D DVRIPE + Dual Radius Polar Wavelet", nullptr, nullptr);
    if(!window){
        std::cerr << "Failed to create GLFW window\n";
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    GLenum glewErr = glewInit();
    if(glewErr != GLEW_OK){
        std::cerr << "GLEW error: " << glewGetErrorString(glewErr) << std::endl;
        glfwDestroyWindow(window);
        glfwTerminate();
        return -1;
    }

    // Allocate 3D field on host
    int n = NX*NY*NZ;
    std::vector<Complex3D> psiHost(n), psiNewHost(n);
    initializeField3D(psiHost, NX, NY, NZ, DX);

    // Allocate on device
    Complex3D* psiDev;
    Complex3D* psiNewDev;
    hipMalloc(&psiDev,    n*sizeof(Complex3D));
    hipMalloc(&psiNewDev, n*sizeof(Complex3D));
    hipMemcpy(psiDev, psiHost.data(), n*sizeof(Complex3D), hipMemcpyHostToDevice);

    // Create PBO for PDE slice
    if(!createPBO(NX, NY)){
        std::cerr << "Failed to create PBO\n";
        return -1;
    }

    // Create 2 wavelet textures for R1 and R2
    if(!createWaveletTexture(waveletTex1, NUM_ANGLES, NUM_SCALES)){
        std::cerr << "Failed to create waveletTex1\n";
        return -1;
    }
    if(!createWaveletTexture(waveletTex2, NUM_ANGLES, NUM_SCALES)){
        std::cerr << "Failed to create waveletTex2\n";
        return -1;
    }

    // PDE kernel config
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((NX+threadsPerBlock.x-1)/threadsPerBlock.x,
                   (NY+threadsPerBlock.y-1)/threadsPerBlock.y,
                   (NZ+threadsPerBlock.z-1)/threadsPerBlock.z);

    // copy kernel config
    int copyThreads=256;
    int copyBlocks=(n+copyThreads-1)/copyThreads;

    // slice kernel config
    dim3 sliceThreads(16,16);
    dim3 sliceBlocks((NX+sliceThreads.x-1)/sliceThreads.x,
                     (NY+sliceThreads.y-1)/sliceThreads.y);

    // Buffers for wavelet
    std::vector<float> angleSignalR1(NUM_ANGLES);
    std::vector<float> angleSignalR2(NUM_ANGLES);

    std::vector<float> waveletScalogramR1(NUM_SCALES*NUM_ANGLES);
    std::vector<float> waveletScalogramR2(NUM_SCALES*NUM_ANGLES);

    while(!glfwWindowShouldClose(window))
    {
        // 1) Evolve PDE
        for(int step=0; step<STEPS_PER_FRAME; step++)
        {
            pdeUpdateKernel3D<<<numBlocks, threadsPerBlock>>>(psiDev, psiNewDev,
                                                              NX, NY, NZ,
                                                              DX, DT, D, G);
            hipDeviceSynchronize();
            copyKernel3D<<<copyBlocks, copyThreads>>>(psiDev, psiNewDev, n);
            hipDeviceSynchronize();
        }

        // 2) Copy to host for wavelet
        hipMemcpy(psiHost.data(), psiDev, n*sizeof(Complex3D), hipMemcpyDeviceToHost);

        // 3) Sample amplitude around R1, R2
        sampleAmplitudeCircle(psiHost, NX, NY, NZ, SLICE_Z,
                              NX/2.0f, NY/2.0f, R1, NUM_ANGLES, angleSignalR1);
        sampleAmplitudeCircle(psiHost, NX, NY, NZ, SLICE_Z,
                              NX/2.0f, NY/2.0f, R2, NUM_ANGLES, angleSignalR2);

        // 4) Morlet wavelet in angle for each radius
        computeMorletWavelet1D(angleSignalR1, NUM_ANGLES, NUM_SCALES,
                               1.0f, 60.0f, W0, waveletScalogramR1);
        computeMorletWavelet1D(angleSignalR2, NUM_ANGLES, NUM_SCALES,
                               1.0f, 60.0f, W0, waveletScalogramR2);

        // 5) Update wavelet textures
        updateWaveletTexture(waveletTex1, waveletScalogramR1, NUM_SCALES, NUM_ANGLES);
        updateWaveletTexture(waveletTex2, waveletScalogramR2, NUM_SCALES, NUM_ANGLES);

        // 6) Map PBO + fill slice
        hipGraphicsMapResources(1, &pboResource, 0);
        size_t numBytes=0;
        uchar4* d_pbo=nullptr;
        hipGraphicsResourceGetMappedPointer((void**)&d_pbo, &numBytes, pboResource);

        fillSliceKernel<<<sliceBlocks, sliceThreads>>>(d_pbo, psiDev,
                                                       NX, NY, NZ, SLICE_Z);
        hipDeviceSynchronize();
        hipGraphicsUnmapResources(1, &pboResource, 0);

        // 7) Render
        glClear(GL_COLOR_BUFFER_BIT);

        // left panel: PDE slice
        {
            glViewport(0, 0, NX, NY);
            glMatrixMode(GL_PROJECTION);
            glLoadIdentity();
            glOrtho(0, NX, 0, NY, -1, 1);
            glMatrixMode(GL_MODELVIEW);
            glLoadIdentity();

            glRasterPos2i(0,0);
            glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboID);
            glDrawPixels(NX, NY, GL_RGBA, GL_UNSIGNED_BYTE, 0);
            glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
        }

        // center panel: wavelet R1
        {
            int panelX = NX; // start at x=NX
            drawTexturedQuad(waveletTex1,
                             panelX, 0,
                             NX, NY,
                             winW, winH);
        }

        // right panel: wavelet R2
        {
            int panelX = NX*2; // start at x=2*NX
            drawTexturedQuad(waveletTex2,
                             panelX, 0,
                             NX, NY,
                             winW, winH);
        }

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // Cleanup
    hipGraphicsUnregisterResource(pboResource);
    glDeleteBuffers(1, &pboID);
    glDeleteTextures(1, &waveletTex1);
    glDeleteTextures(1, &waveletTex2);

    hipFree(psiDev);
    hipFree(psiNewDev);

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
