#include "hip/hip_runtime.h"
/******************************************************************************
 * dvripe_sim_3d_ssf_single_slice.cu
 *
 * Demonstrates a 3D Split-Step Fourier PDE but samples ONLY ONE SLICE (z=0.5)
 * in the 3D texture for debugging. No raymarch.
 *
 * Steps:
 *   1) PDE evolves wavefunction in 32³ domain using split-step Fourier.
 *   2) Each frame, we compute amplitude and upload to a 3D texture.
 *   3) The fragment shader samples the 3D texture at z=0.5 in texture coords
 *      (range [0..1]) and returns a grayscale color for that single slice.
 *   4) Depth testing is DISABLED so we can see the slice even if the cube is
 *      viewed edge-on.
 *
 * Build on Windows (example):
 *   nvcc dvripe_sim_3d_ssf_single_slice.cu -o dvripe_sim_3d_ssf_single_slice.exe ^
        -I"C:\vcpkg\installed\x64-windows\include" ^
        -L"C:\vcpkg\installed\x64-windows\lib" ^
        -lcufft -lglew32 -lglfw3dll -lopengl32
 *
 * Build on Linux (example):
 *   nvcc dvripe_sim_3d_ssf_single_slice.cu -o dvripe_sim_3d_ssf_single_slice \
 *       -lcufft -lGLEW -lglfw -lGL
 ******************************************************************************/

#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// For M_PI on MSVC
#define _USE_MATH_DEFINES
#include <math.h>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// ---------- OpenGL / GLFW / GLEW ----------
#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

//Domadin size
static const int NX = 64;
static const int NY = 64;
static const int NZ = 64;

// PDE parameters
static const float DX   = 0.1f;
static const float DT   = 0.0001f;  // smaller dt => more stable
static const float DVAL = 1.0f;
static const float GVAL = 0.01f;     // weaker focusing => less blow-up

// Steps of PDE per frame
static const int STEPS_PER_FRAME = 1;

// We'll store wavefunction in float2
__device__ inline float2 cMul(const float2& a, const float2& b)
{
    return make_float2(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x);
}
__device__ inline float cAbs2(const float2& a)
{
    return a.x*a.x + a.y*a.y;
}
__device__ inline float2 cExp(float phase)
{
    return make_float2(cosf(phase), sinf(phase));
}

// Nonlinear half-step
__global__ void nonlinearHalfStepKernel(float2* psi, int n, float dt, float G)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n)
    {
        float2 val = psi[idx];
        float amp2 = cAbs2(val);
        float phase = G * amp2 * (dt*0.5f);
        float2 e = make_float2(cosf(phase), sinf(phase));
        float2 out;
        out.x = val.x*e.x - val.y*e.y;
        out.y = val.x*e.y + val.y*e.x;
        psi[idx] = out;
    }
}

// Linear step in Fourier space
__global__ void linearStepKernel(float2* psi_hat,
                                 int nx, int ny, int nz,
                                 float dt, float D,
                                 float dkx, float dky, float dkz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < nx*ny*nz)
    {
        int z = idx / (nx*ny);
        int r = idx % (nx*ny);
        int y = r / nx;
        int x = r % nx;

        // wave numbers
        int sx = (x < nx/2) ? x : (x - nx);
        int sy = (y < ny/2) ? y : (y - ny);
        int sz = (z < nz/2) ? z : (z - nz);

        float kx = sx*dkx;
        float ky = sy*dky;
        float kz = sz*dkz;
        float k2 = kx*kx + ky*ky + kz*kz;

        float phase = -D*k2*dt;
        float2 e = make_float2(cosf(phase), sinf(phase));

        float2 val = psi_hat[idx];
        float2 out;
        out.x = val.x*e.x - val.y*e.y;
        out.y = val.x*e.y + val.y*e.x;
        psi_hat[idx] = out;
    }
}

// Scale after inverse FFT
__global__ void scaleKernel(float2* data, float s, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n)
    {
        data[idx].x *= s;
        data[idx].y *= s;
    }
}

// Compute amplitude
__global__ void computeAmplitudeKernel(const float2* psi, float* amplitude,
                                       int nx, int ny, int nz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < nx*ny*nz)
    {
        float2 val = psi[idx];
        float amp = sqrtf(val.x*val.x + val.y*val.y);
        amplitude[idx] = amp;
    }
}

// Host init
static void initializePsiHost(std::vector<float2>& psiHost,
                              int nx, int ny, int nz, float dx)
{
    float cx = nx/2.0f;
    float cy = ny/2.0f;
    float cz = nz/2.0f;
    for(int z=0; z<nz; z++){
        for(int y=0; y<ny; y++){
            for(int x=0; x<nx; x++){
                int idx = (z*ny + y)*nx + x;
                float dx_ = x - cx;
                float dy_ = y - cy;
                float dz_ = z - cz;
                float r2 = dx_*dx_ + dy_*dy_ + dz_*dz_;

                float amp = expf(-r2/(nx*dx*0.3f));
                // swirl
                float theta = atan2f(dy_, dx_);
                float re = amp*cosf(theta);
                float im = amp*sinf(theta);
                psiHost[idx] = make_float2(re, im);
            }
        }
    }
}

// We'll do a bounding box in [-0.5..0.5]^3
// BUT we will DISABLE DEPTH TEST and sample only z=0.5
// in texture coords in the fragment shader.

// Minimal vertex shader
static const char* vertexShaderSource = R"(
#version 330 core
layout (location = 0) in vec3 inPos;
out vec3 fragPos;
uniform mat4 uMVP;
void main()
{
    fragPos = inPos; // pass to fragment
    gl_Position = uMVP * vec4(inPos, 1.0);
}
)";

// Fragment shader: sample a SINGLE slice at z=0.5 in texture coords
// ignoring the actual z of fragPos.
static const char* fragmentShaderSource = R"(
#version 330 core
in vec3 fragPos;
out vec4 outColor;

uniform sampler3D uVolume;
uniform vec3 uEyePos;

void main()
{
    // We'll do a single slice at z=0.5 in texture coords
    // transform fragPos from [-0.5..+0.5] => [0..1] for x,y
    // but fix z=0.5
    float xCoord = fragPos.x + 0.5;
    float yCoord = fragPos.y + 0.5;

    // if out of [0..1], black
    if(xCoord < 0.0 || xCoord > 1.0 ||
       yCoord < 0.0 || yCoord > 1.0)
    {
        outColor = vec4(0,0,0,1);
        return;
    }

    // z=0.5 in texture coords
    float zCoord = 0.5;

    float amp = texture(uVolume, vec3(xCoord, yCoord, zCoord)).r;
    outColor = vec4(amp, amp, amp, 1.0);
}
)";

static GLuint compileShader(GLenum type, const char* src)
{
    GLuint shader = glCreateShader(type);
    glShaderSource(shader, 1, &src, nullptr);
    glCompileShader(shader);
    GLint success;
    glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
    if(!success){
        char log[1024];
        glGetShaderInfoLog(shader, 1024, nullptr, log);
        std::cerr << "Shader compile error:\n" << log << std::endl;
    }
    return shader;
}

// We'll define a 36-vertex cube, but we'll disable depth test so we always see the front triangles
static GLuint cubeVAO=0, cubeVBO=0;
static void createCube()
{
    float vertices[] = {
        // same old 36-vertex cube from [-0.5..+0.5]^3
        -0.5f, -0.5f,  0.5f,
         0.5f, -0.5f,  0.5f,
         0.5f,  0.5f,  0.5f,

         0.5f,  0.5f,  0.5f,
        -0.5f,  0.5f,  0.5f,
        -0.5f, -0.5f,  0.5f,

        -0.5f, -0.5f, -0.5f,
         0.5f,  0.5f, -0.5f,
         0.5f, -0.5f, -0.5f,

         0.5f,  0.5f, -0.5f,
        -0.5f, -0.5f, -0.5f,
        -0.5f,  0.5f, -0.5f,

        -0.5f,  0.5f,  0.5f,
        -0.5f,  0.5f, -0.5f,
        -0.5f, -0.5f, -0.5f,

        -0.5f, -0.5f, -0.5f,
        -0.5f, -0.5f,  0.5f,
        -0.5f,  0.5f,  0.5f,

         0.5f,  0.5f,  0.5f,
         0.5f, -0.5f, -0.5f,
         0.5f,  0.5f, -0.5f,

         0.5f, -0.5f, -0.5f,
         0.5f,  0.5f,  0.5f,
         0.5f, -0.5f,  0.5f,

        -0.5f,  0.5f,  0.5f,
         0.5f,  0.5f,  0.5f,
         0.5f,  0.5f, -0.5f,

         0.5f,  0.5f, -0.5f,
        -0.5f,  0.5f, -0.5f,
        -0.5f,  0.5f,  0.5f,

        -0.5f, -0.5f,  0.5f,
         0.5f, -0.5f, -0.5f,
         0.5f, -0.5f,  0.5f,

         0.5f, -0.5f, -0.5f,
        -0.5f, -0.5f,  0.5f,
        -0.5f, -0.5f, -0.5f
    };

    glGenVertexArrays(1, &cubeVAO);
    glBindVertexArray(cubeVAO);
    glGenBuffers(1, &cubeVBO);
    glBindBuffer(GL_ARRAY_BUFFER, cubeVBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3*sizeof(float), (void*)0);

    glBindVertexArray(0);
}

static GLuint amplitudeTex=0;
static GLuint prog=0;

static GLuint createShaderProgram()
{
    GLuint vs = compileShader(GL_VERTEX_SHADER, vertexShaderSource);
    GLuint fs = compileShader(GL_FRAGMENT_SHADER, fragmentShaderSource);
    GLuint sp = glCreateProgram();
    glAttachShader(sp, vs);
    glAttachShader(sp, fs);
    glLinkProgram(sp);

    GLint success;
    glGetProgramiv(sp, GL_LINK_STATUS, &success);
    if(!success){
        char log[1024];
        glGetProgramInfoLog(sp, 1024, nullptr, log);
        std::cerr << "Program link error:\n" << log << std::endl;
    }
    glDeleteShader(vs);
    glDeleteShader(fs);
    return sp;
}

static void upload3DTexture(const float* data, int nx, int ny, int nz)
{
    glBindTexture(GL_TEXTURE_3D, amplitudeTex);
    glTexSubImage3D(GL_TEXTURE_3D, 0, 0, 0, 0, nx, ny, nz,
                    GL_RED, GL_FLOAT, data);
    glBindTexture(GL_TEXTURE_3D, 0);
}

int main()
{
    // 1) Init GLFW
    if(!glfwInit()){
        std::cerr << "Failed to init GLFW\n";
        return -1;
    }
    int winW=800, winH=600;
    GLFWwindow* window = glfwCreateWindow(winW, winH, "DVRIPE 3D Single Slice Debug", nullptr, nullptr);
    if(!window){
        std::cerr << "Failed to create GLFW window\n";
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    // 2) GLEW
    GLenum glewErr = glewInit();
    if(glewErr != GLEW_OK){
        std::cerr << "GLEW error: " << glewGetErrorString(glewErr) << std::endl;
        glfwDestroyWindow(window);
        glfwTerminate();
        return -1;
    }

    // 3) PDE setup
    int n = NX*NY*NZ;
    std::vector<float2> psiHost(n);
    initializePsiHost(psiHost, NX, NY, NZ, DX);

    float2* psiDev;
    hipMalloc(&psiDev, n*sizeof(float2));
    hipMemcpy(psiDev, psiHost.data(), n*sizeof(float2), hipMemcpyHostToDevice);

    // cuFFT
    hipfftHandle plan;
    if(hipfftPlan3d(&plan, NZ, NY, NX, HIPFFT_C2C) != HIPFFT_SUCCESS){
        std::cerr << "hipfftPlan3d failed!\n";
        return -1;
    }

    float dkx = 2.0f*M_PI/(NX*DX);
    float dky = 2.0f*M_PI/(NY*DX);
    float dkz = 2.0f*M_PI/(NZ*DX);

    // amplitude device array
    float* ampDev;
    hipMalloc(&ampDev, n*sizeof(float));

    // 4) Create the 3D texture
    glGenTextures(1, &amplitudeTex);
    glBindTexture(GL_TEXTURE_3D, amplitudeTex);
    glTexImage3D(GL_TEXTURE_3D, 0, GL_R32F, NX, NY, NZ, 0,
                 GL_RED, GL_FLOAT, nullptr);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);

    // 5) Create bounding box + shader
    createCube();
    prog = createShaderProgram();

    // pass sampler
    glUseProgram(prog);
    GLint locVol = glGetUniformLocation(prog,"uVolume");
    glUniform1i(locVol, 0);
    glUseProgram(0);

    // PDE kernel config
    int blockSize=256;
    int gridSize=(n+blockSize-1)/blockSize;

    // We'll do a simple orbit camera
    float angleY=0.0f;

    // DISABLE depth test so we see the slice
    glDisable(GL_DEPTH_TEST);

    while(!glfwWindowShouldClose(window))
    {
        // PDE steps
        for(int step=0; step<STEPS_PER_FRAME; step++)
        {
            // half-step
            nonlinearHalfStepKernel<<<gridSize, blockSize>>>(psiDev, n, DT, GVAL);
            hipDeviceSynchronize();

            // forward FFT
            hipfftExecC2C(plan, (hipfftComplex*)psiDev, (hipfftComplex*)psiDev, HIPFFT_FORWARD);

            // linear
            linearStepKernel<<<gridSize, blockSize>>>(psiDev, NX, NY, NZ, DT, DVAL, dkx, dky, dkz);
            hipDeviceSynchronize();

            // inverse FFT
            hipfftExecC2C(plan, (hipfftComplex*)psiDev, (hipfftComplex*)psiDev, HIPFFT_BACKWARD);

            // scale
            float scale = 1.0f/(float)(NX*NY*NZ);
            scaleKernel<<<gridSize, blockSize>>>(psiDev, scale, n);
            hipDeviceSynchronize();

            // half-step
            nonlinearHalfStepKernel<<<gridSize, blockSize>>>(psiDev, n, DT, GVAL);
            hipDeviceSynchronize();
        }

        // compute amplitude
        computeAmplitudeKernel<<<gridSize, blockSize>>>(psiDev, ampDev, NX, NY, NZ);
        hipDeviceSynchronize();

        // copy to host
        std::vector<float> ampHost(n);
        hipMemcpy(ampHost.data(), ampDev, n*sizeof(float), hipMemcpyDeviceToHost);

        // upload
        upload3DTexture(ampHost.data(), NX, NY, NZ);

        // revolve camera
        angleY += 0.01f;

        glClearColor(0,0,0,1);
        glClear(GL_COLOR_BUFFER_BIT);

        // basic perspective
        float fov=60.0f*(M_PI/180.0f);
        float aspect=(float)winW/(float)winH;
        float nearP=0.1f, farP=10.0f;
        auto matIdentity=[](){
            float m[16]={1,0,0,0, 0,1,0,0, 0,0,1,0, 0,0,0,1};
            return std::vector<float>(m,m+16);
        };
        auto matMultiply=[](const std::vector<float>&A,const std::vector<float>&B){
            std::vector<float> M(16,0);
            for(int r=0;r<4;r++){
                for(int c=0;c<4;c++){
                    for(int k=0;k<4;k++){
                        M[r*4+c]+=A[r*4+k]*B[k*4+c];
                    }
                }
            }
            return M;
        };

        float f = 1.0f/tanf(fov*0.5f);
        std::vector<float> matP(16,0);
        matP[0]=f/aspect; matP[5]=f;
        matP[10]=(farP+nearP)/(nearP-farP);
        matP[11]=-1;
        matP[14]=(2.0f*farP*nearP)/(nearP-farP);

        auto sub3=[](float x1,float y1,float z1,float x2,float y2,float z2){
            return std::vector<float>{x1-x2,y1-y2,z1-z2};
        };
        auto norm3=[](const std::vector<float>& v){
            float l=sqrtf(v[0]*v[0]+v[1]*v[1]+v[2]*v[2]);
            return std::vector<float>{v[0]/l,v[1]/l,v[2]/l};
        };
        auto cross3=[](const std::vector<float>&a,const std::vector<float>&b){
            return std::vector<float>{
                a[1]*b[2] - a[2]*b[1],
                a[2]*b[0] - a[0]*b[2],
                a[0]*b[1] - a[1]*b[0]
            };
        };

        float dist=1.5f;
        float eyex = dist*sinf(angleY);
        float eyey = 0.0f;
        float eyez = dist*cosf(angleY);
        std::vector<float> eye={eyex,eyey,eyez};
        std::vector<float> center={0,0,0};
        std::vector<float> up={0,1,0};

        auto fwd=norm3(sub3(center[0],center[1],center[2], eye[0],eye[1],eye[2]));
        auto rht=norm3(cross3(fwd, up));
        auto u2 = cross3(rht, fwd);

        std::vector<float> matV=matIdentity();
        matV[0]=rht[0]; matV[1]=u2[0]; matV[2]=-fwd[0];
        matV[4]=rht[1]; matV[5]=u2[1]; matV[6]=-fwd[1];
        matV[8]=rht[2]; matV[9]=u2[2]; matV[10]=-fwd[2];

        float tx=-(rht[0]*eye[0]+rht[1]*eye[1]+rht[2]*eye[2]);
        float ty=-(u2[0]*eye[0]+u2[1]*eye[1]+u2[2]*eye[2]);
        float tz= (fwd[0]*eye[0]+fwd[1]*eye[1]+fwd[2]*eye[2]);
        matV[12]=tx; matV[13]=ty; matV[14]=tz;

        auto matM=matIdentity();
        auto matVM=matMultiply(matV, matM);
        auto matPVM=matMultiply(matP, matVM);

        glUseProgram(prog);
        GLint locMVP= glGetUniformLocation(prog,"uMVP");
        glUniformMatrix4fv(locMVP,1,GL_FALSE, matPVM.data());

        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_3D, amplitudeTex);

        glBindVertexArray(cubeVAO);
        glDrawArrays(GL_TRIANGLES, 0, 36);
        glBindVertexArray(0);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    hipfftDestroy(plan);
    hipFree(psiDev);
    hipFree(ampDev);

    glDeleteTextures(1, &amplitudeTex);
    glDeleteProgram(prog);
    glDeleteVertexArrays(1, &cubeVAO);
    glDeleteBuffers(1, &cubeVBO);

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
