#include "hip/hip_runtime.h"
/******************************************************************************
 * dvripe_sim_3d_ssf_follow_vortex.cu
 *
 * Demonstrates a 3D Split-Step Fourier integrator for a focusing Schrödinger 
 * equation, then locates the maximum amplitude ("vortex center") each frame and 
 * translates the volume so that point is at the center of the view. We sample 
 * only one slice (z=0.5) in texture space for simplicity.
 *
 * Build on Windows (example):
 *   nvcc dvripe_sim_3d_ssf_follow_vortex.cu -o dvripe_sim_3d_ssf_follow_vortex.exe ^
        -I"C:\vcpkg\installed\x64-windows\include" ^
        -L"C:\vcpkg\installed\x64-windows\lib" ^
        -lcufft -lglew32 -lglfw3dll -lopengl32
 *
 * Build on Linux (example):
 *   nvcc dvripe_sim_3d_ssf_follow_vortex.cu -o dvripe_sim_3d_ssf_follow_vortex \
 *       -lcufft -lGLEW -lglfw -lGL
 *
 * Notes:
 *   - Domain is 64³, G=0.2, DT=0.0005 => moderate focusing. 
 *   - We do a single-slice visualization (z=0.5) to keep the code simpler. 
 *   - We find the maximum amplitude index each frame, and shift the model 
 *     matrix so that point appears at the center in [-0.5..0.5].
 ******************************************************************************/

#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// For M_PI on MSVC
#define _USE_MATH_DEFINES
#include <math.h>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// OpenGL / GLFW / GLEW
#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

// ------------------------------------------
// Simulation grid
static const int NX = 64;
static const int NY = 64;
static const int NZ = 64;

// PDE parameters
static const float DX   = 0.1f;
static const float DT   = 0.0001f;  // smaller dt => more stable
static const float DVAL = 1.0f;
static const float GVAL = 1.0f;     // stronger focusing

// Steps of PDE per frame
static const int STEPS_PER_FRAME = 1;

// ------------------------------------------
// GPU complex math
// ------------------------------------------
__device__ inline float2 cMul(const float2& a, const float2& b)
{
    return make_float2(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x);
}
__device__ inline float cAbs2(const float2& a)
{
    return a.x*a.x + a.y*a.y;
}
__device__ inline float2 cExp(float phase)
{
    return make_float2(cosf(phase), sinf(phase));
}

// ------------------------------------------
// Nonlinear half-step: ψ <- ψ * exp(i G |ψ|^2 dt/2)
// ------------------------------------------
__global__ void nonlinearHalfStepKernel(float2* psi, int n, float dt, float G)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < n)
    {
        float2 val = psi[idx];
        float amp2 = cAbs2(val);
        float phase = G * amp2 * (dt*0.5f);
        float2 e = make_float2(cosf(phase), sinf(phase));
        float2 out;
        out.x = val.x*e.x - val.y*e.y;
        out.y = val.x*e.y + val.y*e.x;
        psi[idx] = out;
    }
}

// ------------------------------------------
// Linear step in Fourier space: multiply each mode by exp(-i D k^2 dt)
// ------------------------------------------
__global__ void linearStepKernel(float2* psi_hat,
                                 int nx, int ny, int nz,
                                 float dt, float D,
                                 float dkx, float dky, float dkz)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < (nx*ny*nz))
    {
        int z = idx / (nx*ny);
        int r = idx % (nx*ny);
        int y = r / nx;
        int x = r % nx;

        // wave numbers
        int sx = (x < nx/2) ? x : (x - nx);
        int sy = (y < ny/2) ? y : (y - ny);
        int sz = (z < nz/2) ? z : (z - nz);

        float kx = sx*dkx;
        float ky = sy*dky;
        float kz = sz*dkz;
        float k2 = kx*kx + ky*ky + kz*kz;

        float phase = -D*k2*dt;
        float2 e = make_float2(cosf(phase), sinf(phase));

        float2 val = psi_hat[idx];
        float2 out;
        out.x = val.x*e.x - val.y*e.y;
        out.y = val.x*e.y + val.y*e.x;
        psi_hat[idx] = out;
    }
}

// ------------------------------------------
// Scale after inverse FFT
// ------------------------------------------
__global__ void scaleKernel(float2* data, float s, int n)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < n)
    {
        data[idx].x *= s;
        data[idx].y *= s;
    }
}

// ------------------------------------------
// Compute amplitude
// ------------------------------------------
__global__ void computeAmplitudeKernel(const float2* psi, float* amplitude,
                                       int nx, int ny, int nz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < nx*ny*nz)
    {
        float2 val = psi[idx];
        float amp = sqrtf(val.x*val.x + val.y*val.y);
        amplitude[idx] = amp;
    }
}

// ------------------------------------------
// Host function to initialize psi with a swirl
// ------------------------------------------
static void initializePsiHost(std::vector<float2>& psiHost,
                              int nx, int ny, int nz, float dx)
{
    float cx = nx/2.0f;
    float cy = ny/2.0f;
    float cz = nz/2.0f;
    for(int z=0; z<nz; z++){
        for(int y=0; y<ny; y++){
            for(int x=0; x<nx; x++){
                int idx = (z*ny + y)*nx + x;
                float dx_ = x - cx;
                float dy_ = y - cy;
                float dz_ = z - cz;
                float r2 = dx_*dx_ + dy_*dy_ + dz_*dz_;

                // amplitude => Gaussian
                float amp = expf(-r2/(nx*dx*0.3f));
                // swirl around z => phase = atan2f(y - centerY, x - centerX)
                float theta = atan2f(dy_, dx_);
                float re = amp*cosf(theta);
                float im = amp*sinf(theta);
                psiHost[idx] = make_float2(re, im);
            }
        }
    }
}

// ------------------------------------------
// We'll do a bounding box from [-0.5..+0.5]^3
// We'll sample a single slice at z=0.5 in the fragment shader
// But we will SHIFT the volume so the vortex peak is at the domain center
// in local coords. We'll do that by a translation in the model matrix.
//
// Vertex shader: pass fragPos
// Fragment shader: single slice
// ------------------------------------------

static const char* vertexShaderSource = R"(
#version 330 core
layout (location = 0) in vec3 inPos;
out vec3 fragPos;
uniform mat4 uMVP;
void main()
{
    fragPos = inPos; 
    gl_Position = uMVP * vec4(inPos, 1.0);
}
)";

// We'll just sample z=0.5 in texture coords, ignoring fragPos.z
static const char* fragmentShaderSource = R"(
#version 330 core
in vec3 fragPos;
out vec4 outColor;

uniform sampler3D uVolume;

// The volume is in [-0.5..+0.5]^3 after our model transform
// We sample xCoord, yCoord in [0..1], zCoord=0.5
void main()
{
    float xCoord = fragPos.x + 0.5;
    float yCoord = fragPos.y + 0.5;
    if(xCoord < 0.0 || xCoord > 1.0 ||
       yCoord < 0.0 || yCoord > 1.0)
    {
        outColor = vec4(0,0,0,1);
        return;
    }
    float zCoord = 0.5;

    float amp = texture(uVolume, vec3(xCoord, yCoord, zCoord)).r;
    outColor = vec4(amp, amp, amp, 1.0);
}
)";

static GLuint compileShader(GLenum type, const char* src)
{
    GLuint shader = glCreateShader(type);
    glShaderSource(shader, 1, &src, nullptr);
    glCompileShader(shader);
    GLint success;
    glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
    if(!success){
        char log[1024];
        glGetShaderInfoLog(shader, 1024, nullptr, log);
        std::cerr << "Shader compile error:\n" << log << std::endl;
    }
    return shader;
}

static GLuint cubeVAO=0, cubeVBO=0;
static void createCube()
{
    float vertices[] = {
        // same 36-vertex cube in [-0.5..+0.5]^3
        -0.5f, -0.5f,  0.5f,
         0.5f, -0.5f,  0.5f,
         0.5f,  0.5f,  0.5f,

         0.5f,  0.5f,  0.5f,
        -0.5f,  0.5f,  0.5f,
        -0.5f, -0.5f,  0.5f,

        -0.5f, -0.5f, -0.5f,
         0.5f,  0.5f, -0.5f,
         0.5f, -0.5f, -0.5f,

         0.5f,  0.5f, -0.5f,
        -0.5f, -0.5f, -0.5f,
        -0.5f,  0.5f, -0.5f,

        -0.5f,  0.5f,  0.5f,
        -0.5f,  0.5f, -0.5f,
        -0.5f, -0.5f, -0.5f,

        -0.5f, -0.5f, -0.5f,
        -0.5f, -0.5f,  0.5f,
        -0.5f,  0.5f,  0.5f,

         0.5f,  0.5f,  0.5f,
         0.5f, -0.5f, -0.5f,
         0.5f,  0.5f, -0.5f,

         0.5f, -0.5f, -0.5f,
         0.5f, -0.5f,  0.5f,
         0.5f,  0.5f,  0.5f,

        -0.5f,  0.5f,  0.5f,
         0.5f,  0.5f,  0.5f,
         0.5f,  0.5f, -0.5f,

         0.5f,  0.5f, -0.5f,
        -0.5f,  0.5f, -0.5f,
        -0.5f,  0.5f,  0.5f,

        -0.5f, -0.5f,  0.5f,
         0.5f, -0.5f, -0.5f,
         0.5f, -0.5f,  0.5f,

         0.5f, -0.5f, -0.5f,
        -0.5f, -0.5f,  0.5f,
        -0.5f, -0.5f, -0.5f
    };

    glGenVertexArrays(1, &cubeVAO);
    glBindVertexArray(cubeVAO);
    glGenBuffers(1, &cubeVBO);
    glBindBuffer(GL_ARRAY_BUFFER, cubeVBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3*sizeof(float), (void*)0);

    glBindVertexArray(0);
}

static GLuint amplitudeTex=0;
static GLuint prog=0;

static GLuint createShaderProgram()
{
    GLuint vs = compileShader(GL_VERTEX_SHADER, vertexShaderSource);
    GLuint fs = compileShader(GL_FRAGMENT_SHADER, fragmentShaderSource);
    GLuint sp = glCreateProgram();
    glAttachShader(sp, vs);
    glAttachShader(sp, fs);
    glLinkProgram(sp);

    GLint success;
    glGetProgramiv(sp, GL_LINK_STATUS, &success);
    if(!success){
        char log[1024];
        glGetProgramInfoLog(sp, 1024, nullptr, log);
        std::cerr << "Program link error:\n" << log << std::endl;
    }
    glDeleteShader(vs);
    glDeleteShader(fs);
    return sp;
}

// We'll upload amplitude to the 3D texture
static void upload3DTexture(const float* data, int nx, int ny, int nz)
{
    glBindTexture(GL_TEXTURE_3D, amplitudeTex);
    glTexSubImage3D(GL_TEXTURE_3D, 0, 0, 0, 0, nx, ny, nz,
                    GL_RED, GL_FLOAT, data);
    glBindTexture(GL_TEXTURE_3D, 0);
}

// Find the maximum amplitude index on the host
// Return (peakX, peakY, peakZ)
static void findPeakAmplitude(const std::vector<float>& ampHost,
                              int nx, int ny, int nz,
                              int& peakX, int& peakY, int& peakZ)
{
    float maxVal=0.0f;
    peakX=0; peakY=0; peakZ=0;
    for(int z=0; z<nz; z++){
        for(int y=0; y<ny; y++){
            for(int x=0; x<nx; x++){
                int idx=(z*ny + y)*nx + x;
                float val=ampHost[idx];
                if(val>maxVal){
                    maxVal=val;
                    peakX=x; peakY=y; peakZ=z;
                }
            }
        }
    }
}

int main()
{
    // 1) Init GLFW
    if(!glfwInit()){
        std::cerr << "Failed to init GLFW\n";
        return -1;
    }
    int winW=800, winH=600;
    GLFWwindow* window = glfwCreateWindow(winW, winH, "DVRIPE 3D Follow Vortex", nullptr, nullptr);
    if(!window){
        std::cerr << "Failed to create GLFW window\n";
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    // 2) GLEW
    GLenum glewErr = glewInit();
    if(glewErr != GLEW_OK){
        std::cerr << "GLEW error: " << glewGetErrorString(glewErr) << std::endl;
        glfwDestroyWindow(window);
        glfwTerminate();
        return -1;
    }

    // 3) PDE setup
    int n = NX*NY*NZ;
    std::vector<float2> psiHost(n);
    initializePsiHost(psiHost, NX, NY, NZ, DX);

    float2* psiDev;
    hipMalloc(&psiDev, n*sizeof(float2));
    hipMemcpy(psiDev, psiHost.data(), n*sizeof(float2), hipMemcpyHostToDevice);

    // cuFFT
    hipfftHandle plan;
    if(hipfftPlan3d(&plan, NZ, NY, NX, HIPFFT_C2C) != HIPFFT_SUCCESS){
        std::cerr << "hipfftPlan3d failed!\n";
        return -1;
    }

    float dkx = 2.0f*M_PI/(NX*DX);
    float dky = 2.0f*M_PI/(NY*DX);
    float dkz = 2.0f*M_PI/(NZ*DX);

    // amplitude device array
    float* ampDev;
    hipMalloc(&ampDev, n*sizeof(float));

    // 4) Create the 3D texture
    glGenTextures(1, &amplitudeTex);
    glBindTexture(GL_TEXTURE_3D, amplitudeTex);
    glTexImage3D(GL_TEXTURE_3D, 0, GL_R32F, NX, NY, NZ, 0,
                 GL_RED, GL_FLOAT, nullptr);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);

    // 5) Create bounding box + shader
    createCube();
    prog = createShaderProgram();

    glUseProgram(prog);
    GLint locVol = glGetUniformLocation(prog,"uVolume");
    glUniform1i(locVol, 0);
    glUseProgram(0);

    // PDE kernel config
    int blockSize=256;
    int gridSize=(n+blockSize-1)/blockSize;

    // We'll do a simple orbit camera
    float angleY=0.0f;

    // disable depth so we see the slice
    glDisable(GL_DEPTH_TEST);

    while(!glfwWindowShouldClose(window))
    {
        // PDE steps
        for(int step=0; step<STEPS_PER_FRAME; step++)
        {
            // half-step
            nonlinearHalfStepKernel<<<gridSize, blockSize>>>(psiDev, n, DT, GVAL);
            hipDeviceSynchronize();

            // forward FFT
            hipfftExecC2C(plan, (hipfftComplex*)psiDev, (hipfftComplex*)psiDev, HIPFFT_FORWARD);

            // linear
            linearStepKernel<<<gridSize, blockSize>>>(psiDev, NX, NY, NZ, DT, DVAL, dkx, dky, dkz);
            hipDeviceSynchronize();

            // inverse FFT
            hipfftExecC2C(plan, (hipfftComplex*)psiDev, (hipfftComplex*)psiDev, HIPFFT_BACKWARD);

            // scale
            float scale = 1.0f/(float)(NX*NY*NZ);
            scaleKernel<<<gridSize, blockSize>>>(psiDev, scale, n);
            hipDeviceSynchronize();

            // half-step
            nonlinearHalfStepKernel<<<gridSize, blockSize>>>(psiDev, n, DT, GVAL);
            hipDeviceSynchronize();
        }

        // compute amplitude
        computeAmplitudeKernel<<<gridSize, blockSize>>>(psiDev, ampDev, NX, NY, NZ);
        hipDeviceSynchronize();

        // copy to host
        std::vector<float> ampHost(n);
        hipMemcpy(ampHost.data(), ampDev, n*sizeof(float), hipMemcpyDeviceToHost);

        // find peak amplitude index
        int peakX=0, peakY=0, peakZ=0;
        findPeakAmplitude(ampHost, NX, NY, NZ, peakX, peakY, peakZ);

        // convert that index to a shift in [-0.5..+0.5]
        // i in [0..NX-1], so normalizedCoord = (i + 0.5f)/NX
        float fx = (peakX + 0.5f)/(float)NX;
        float fy = (peakY + 0.5f)/(float)NY;
        float fz = (peakZ + 0.5f)/(float)NZ;
        // shift so that (fx, fy, fz) => (0.5, 0.5, 0.5)
        // means we want a translation of (0.5 - fx, etc.)
        float shiftX = 0.5f - fx;
        float shiftY = 0.5f - fy;
        float shiftZ = 0.5f - fz;

        // upload amplitude to 3D texture
        upload3DTexture(ampHost.data(), NX, NY, NZ);

        // revolve camera
        angleY += 0.01f;

        glClearColor(0,0,0,1);
        glClear(GL_COLOR_BUFFER_BIT);

        // basic perspective
        float fov=60.0f*(M_PI/180.0f);
        float aspect=(float)winW/(float)winH;
        float nearP=0.1f, farP=10.0f;
        auto matIdentity=[](){
            float m[16]={1,0,0,0, 0,1,0,0, 0,0,1,0, 0,0,0,1};
            return std::vector<float>(m,m+16);
        };
        auto matMultiply=[](const std::vector<float>&A,const std::vector<float>&B){
            std::vector<float> M(16,0);
            for(int r=0;r<4;r++){
                for(int c=0;c<4;c++){
                    for(int k=0;k<4;k++){
                        M[r*4+c]+=A[r*4+k]*B[k*4+c];
                    }
                }
            }
            return M;
        };

        float f = 1.0f/tanf(fov*0.5f);
        std::vector<float> matP(16,0);
        matP[0]=f/aspect; matP[5]=f;
        matP[10]=(farP+nearP)/(nearP-farP);
        matP[11]=-1;
        matP[14]=(2.0f*farP*nearP)/(nearP-farP);

        // minimal orbit camera
        auto sub3=[](float x1,float y1,float z1,float x2,float y2,float z2){
            return std::vector<float>{x1-x2,y1-y2,z1-z2};
        };
        auto norm3=[](const std::vector<float>& v){
            float l=sqrtf(v[0]*v[0]+v[1]*v[1]+v[2]*v[2]);
            return std::vector<float>{v[0]/l,v[1]/l,v[2]/l};
        };
        auto cross3=[](const std::vector<float>&a,const std::vector<float>&b){
            return std::vector<float>{
                a[1]*b[2] - a[2]*b[1],
                a[2]*b[0] - a[0]*b[2],
                a[0]*b[1] - a[1]*b[0]
            };
        };

        float dist=1.5f;
        float eyex = dist*sinf(angleY);
        float eyey = 0.0f;
        float eyez = dist*cosf(angleY);
        std::vector<float> eye={eyex,eyey,eyez};
        std::vector<float> center={0,0,0};
        std::vector<float> up={0,1,0};

        auto fwd=norm3(sub3(center[0],center[1],center[2], eye[0],eye[1],eye[2]));
        auto rht=norm3(cross3(fwd, up));
        auto u2 = cross3(rht, fwd);

        std::vector<float> matV=matIdentity();
        matV[0]=rht[0]; matV[1]=u2[0]; matV[2]=-fwd[0];
        matV[4]=rht[1]; matV[5]=u2[1]; matV[6]=-fwd[1];
        matV[8]=rht[2]; matV[9]=u2[2]; matV[10]=-fwd[2];

        float tx=-(rht[0]*eye[0]+rht[1]*eye[1]+rht[2]*eye[2]);
        float ty=-(u2[0]*eye[0]+u2[1]*eye[1]+u2[2]*eye[2]);
        float tz= (fwd[0]*eye[0]+fwd[1]*eye[1]+fwd[2]*eye[2]);
        matV[12]=tx; matV[13]=ty; matV[14]=tz;

        // model transform => shift volume by (shiftX, shiftY, shiftZ)
        // so that the vortex peak is at (0.5, 0.5, 0.5) in local coords
        // => [-0.5..+0.5]^3 => means we do a translation of shift?
        // We define a minimal translation matrix:
        auto matTranslate=[&](float sx,float sy,float sz){
            std::vector<float> T=matIdentity();
            T[12]=sx; T[13]=sy; T[14]=sz;
            return T;
        };
        auto matM=matTranslate(shiftX, shiftY, shiftZ);

        // combine
        auto matVM=matMultiply(matV, matM);
        auto matPVM=matMultiply(matP, matVM);

        glClear(GL_COLOR_BUFFER_BIT);

        glUseProgram(prog);
        GLint locMVP= glGetUniformLocation(prog,"uMVP");
        glUniformMatrix4fv(locMVP,1,GL_FALSE, matPVM.data());

        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_3D, amplitudeTex);

        glBindVertexArray(cubeVAO);
        glDrawArrays(GL_TRIANGLES, 0, 36);
        glBindVertexArray(0);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    hipfftDestroy(plan);
    hipFree(psiDev);
    hipFree(ampDev);

    glDeleteTextures(1, &amplitudeTex);
    glDeleteProgram(prog);
    glDeleteVertexArrays(1, &cubeVAO);
    glDeleteBuffers(1, &cubeVBO);

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
